// nvcc -o Cuda_pi Cuda_pi.cu
// Run Unix: ./Cuda_pi

#include <stdio.h>
#include <math.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define BLOCKSPERGRID  512
#define NUMTHREADS 8192
#define ITERATIONS 16e09

__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads) {
    long int initIteration, endIteration;
    long int i = 0;
    double piPartial = 0.0;
    
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;

    initIteration = (iterations / totalThreads) * index;
    endIteration = initIteration + (iterations / totalThreads) - 1;
    
    i = initIteration;
    
    do {
        piPartial = piPartial + (double)(4.0 / ((i * 2) + 1));
        i++;
        piPartial = piPartial - (double)(4.0 / ((i * 2) + 1));
        i++;
    } while(i < endIteration);

    piTotal[index] = piPartial;
    
    __syncthreads();

    if(index == 0) {
        for(i = 1; i < totalThreads; i++) {
            piTotal[0] = piTotal[0] + piTotal[i];
        }
    }
}

int main() {
    int blocksPerGrid, threadsPerBlock, i, size;
    long int iterations;
    int totalThreads;
    double *h_pitotal, *d_pitotal;
    
    blocksPerGrid = BLOCKSPERGRID;
    hipError_t err = hipSuccess;

    size = sizeof(double)*NUMTHREADS;
    h_pitotal = (double *)malloc(size);

    if (h_pitotal == NULL){
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    
    for(i = 0; i < NUMTHREADS; i++) {
        h_pitotal[i] = 0.0;
    }

    err = hipMalloc((void **)&d_pitotal, size);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_pitotal, h_pitotal, sizeof(double)*NUMTHREADS, hipMemcpyHostToDevice);

    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Lanzar KERNEL
    threadsPerBlock = NUMTHREADS/blocksPerGrid;
    totalThreads = blocksPerGrid * threadsPerBlock;
    iterations = ITERATIONS;
    printf("CUDA kernel launch with %d blocks of %d threads Total: %i       ", blocksPerGrid, threadsPerBlock, totalThreads  );
    calculatePi<<<blocksPerGrid, threadsPerBlock>>>(d_pitotal, iterations, totalThreads);
    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_pitotal, d_pitotal, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_pitotal);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Calculated pi: %.12f", *h_pitotal);
    // Free host memory

    free(h_pitotal);
    err = hipDeviceReset();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}